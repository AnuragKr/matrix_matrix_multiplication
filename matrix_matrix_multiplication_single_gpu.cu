#include<stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

__global__ void matMulGpu(int *a,int *b,int *c,int matrixsize){
        int row = blockIdx.x;
        int col = threadIdx.x;

        for(int k=0;k<blockDim.x;k++)
                c[row*matrixsize+col] += a[row*matrixsize+k]*b[k*matrixsize+col];
}

//Utility function for time calculation
double rtclock() {
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) printf("Error return from gettimeofday: %d", stat);
  return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printTime(const char *str, double starttime, double endtime) {
	printf("%s%3f seconds\n", str, endtime - starttime);
}

#define N 1024
#define BLOCKSIZE N
int main(){
    //Declaring Variables for Host
    int *a,*b,*c;

    //Declaring Variables for Device
    int *d_a,*d_b,*d_c,*d_d;

    //Allocated memory for variables with array of size N*N  on the Host
    a=(int *)malloc(sizeof(int)*N*N);
    b=(int *)malloc(sizeof(int)*N*N);
    c=(int *)malloc(sizeof(int)*N*N);

    //Allocated memory for variables with array of size N*N on the Device
    if(hipMalloc((void **)&d_a,sizeof(int)*N*N)!=hipSuccess) printf("Error in allocation d_a \n");
    if(hipMalloc((void **)&d_b,sizeof(int)*N*N)!=hipSuccess) printf("Error in allocation d_b \n");
    if(hipMalloc((void **)&d_c,sizeof(int)*N*N)!=hipSuccess) printf("Error in allocation d_c \n");
    if(hipMalloc((void **)&d_d,sizeof(int)*N*N)!=hipSuccess) printf("Error in allocation d_c \n");


    //Initialized Host variables
    for(int i=0;i<N*N;i++)a[i]=b[i]=1;

    //Copying Host variables content to Device
    if(hipMemcpy(d_a,a,N*N*sizeof(int),hipMemcpyHostToDevice) != hipSuccess)printf("memcpy a->d_a failed \n");
    if(hipMemcpy(d_b,b,N*N*sizeof(int),hipMemcpyHostToDevice) != hipSuccess)printf("memcpy b->d_b failed \n");

    //Launch the kernel for computation on the device
    int nblocks = ceil((int)N / BLOCKSIZE);
    double starttime,endtime;

    //GPU-2
    starttime = rtclock();
    matMulGpu<<<nblocks * N, BLOCKSIZE>>>(d_a,d_b,d_d,N);
    //Waiting For GPU Device to Finish computation
    hipDeviceSynchronize();
    endtime = rtclock();
    printTime("GPU time: ", starttime, endtime);
    printf("\n");

    //Copying Result Synchronously from GPU to CPU
    if(hipMemcpy(c,d_d,sizeof(int)*N*N,hipMemcpyDeviceToHost)!=hipSuccess)printf("\nCopy result failed\n");

    //Checking result
    for(int i=0;i<N*N;i++){ 
	    if(c[i]!=N){
		    printf("Error in result\n");
		    break;
	    }
    }

    //Free up the memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);

    return 0;
}
