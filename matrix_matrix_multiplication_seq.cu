#include<stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

void matMulCpu(int *a,int *b,int *c, int matrixsize) {
        for (int i = 0; i < matrixsize; ++i) {
	        for (int j = 0; j < matrixsize; ++j) {
		        for (int k = 0; k < matrixsize; ++k) {
			        c[i * matrixsize + j] += a[i * matrixsize + k] * b[k * matrixsize + j];
		        }
	        }
	}
}

//Utility function for time calculation
double rtclock() {
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) printf("Error return from gettimeofday: %d", stat);
  return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printTime(const char *str, double starttime, double endtime) {
	printf("%s%3f seconds\n", str, endtime - starttime);
}

#define N 1024
#define BLOCKSIZE N
int main(){
    //Declaring Variables for Host
    int *a,*b,*c;

    //Allocated memory for variables with array of size N*N  on the Host
    a=(int *)malloc(sizeof(int)*N*N);
    b=(int *)malloc(sizeof(int)*N*N);
    c=(int *)malloc(sizeof(int)*N*N);

    //Initialized Host variables
    for(int i=0;i<N*N;i++)a[i]=b[i]=1;

    //CPU
    double starttime,endtime;
    starttime = rtclock();
    matMulCpu(a,b,c,N);
    endtime = rtclock();
    printTime("CPU time: ", starttime, endtime);

    return 0;
}
